#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include "argparse.h"
#include "hip/hip_runtime_api.h"

static const char *const usage[] = {
    "device_query [options]",
    NULL,
};

void pad_string(char *des, char *src, size_t target_length);

void print_general_info(hipDeviceProp_t deviceProp);

void print_advanced_features(hipDeviceProp_t deviceProp);

void print_programming_features(hipDeviceProp_t deviceProp);


void print_device_info(int device_id) {
    hipDeviceProp_t deviceProp;
    hipError_t err = hipGetDeviceProperties(&deviceProp, device_id);
    if (err != hipSuccess) {
        printf("fail to query gpu #%d\n", device_id);
        return;
    }

    printf("\nDevice %d\n", device_id);
    print_general_info(deviceProp);
    print_programming_features(deviceProp);
    print_advanced_features(deviceProp);
}

int main(int argc, const char **argv) {
    int device_id = -1;
    struct argparse_option options[] = {
        OPT_HELP(),
        OPT_INTEGER('d', "device", &device_id, "deivce id, default to 0"),
        OPT_END(),
    };
    struct argparse argparse;
    argparse_init(&argparse, options, usage, 0);
    int argc_remain = argparse_parse(&argparse, argc, argv);
    // printf("num of remaining arguments: %d\n", argc);
    // printf("device id: %d\n", device_id);

    int num_devices = 0;
    hipError_t err = hipGetDeviceCount(&num_devices);
    if (err != hipSuccess) {
        printf("fail to query CUDA devices\n");
        return -1;
    }
    if (device_id != -1 && device_id >= num_devices) {
        printf("GPU #%d does not exit: total number of CUDA devices: %d\n", device_id, num_devices);
        return 2;
    }

    int driverVersion = 0, runtimeVersion = 0;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);

    if (device_id == -1) {
        for (device_id = 0; device_id < num_devices; device_id++) {
            print_device_info(device_id);
        }
    } else {
        print_device_info(device_id);
    }

    return 0;
}


void print_general_info(hipDeviceProp_t deviceProp) {
    // name,
    // integrated,
    // major, minor,
    // multiProcessorCount,
    // clockRate,
    // memoryClockRate,
    // memoryBusWidth,
    // l2CacheSize,

    const int description_len = 68;
    char *padded = (char *)malloc(sizeof(char) * (description_len + 1));

    printf("%s (%s), ", deviceProp.name,
           _ConvertSMVer2ArchName(deviceProp.major, deviceProp.minor));
    if (deviceProp.integrated == 1) {
        printf("integrated\n");
    } else {
        printf("decrete\n");
    }

    pad_string(padded, "  CUDA Capability Major/Minor version number:", description_len);
    printf("%s%d.%d\n", padded, deviceProp.major, deviceProp.minor);

    printf("  (%03d) Multiprocessors, (%03d) CUDA Cores/MP:                        %d CUDA Cores\n",
           deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
           deviceProp.multiProcessorCount);

    pad_string(padded, "  GPU Max Clock rate:", description_len);
    printf("%s%.0f MHz (%0.2f GHz)\n", padded, deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

    pad_string(padded, "  Memory Clock rate:", description_len);
    printf("%s%.0f Mhz\n", padded, deviceProp.memoryClockRate * 1e-3f);

    pad_string(padded, "  Memory Bus Width:", description_len);
    printf("%s%d-bit\n", padded, deviceProp.memoryBusWidth);

    pad_string(padded, "  L2 Cache Size:", description_len);
    printf("%s%d bytes\n", padded, deviceProp.l2CacheSize);

    free(padded);
    return;
}

void print_advanced_features(hipDeviceProp_t deviceProp) {
    // computePreemptionSupported,
    // streamPrioritiesSupported,
    // globalL1CacheSupported,
    // localL1CacheSupported,

    // unifiedAddressing,
    // canMapHostMemory,
    // managedMemory,
    // directManagedMemAccessFromHost,
    // asyncEngineCount,

    const int description_len = 68;
    char *padded = (char *)malloc(sizeof(char) * (description_len + 1));

    pad_string(padded, "  Device supports Compute Preemption:", description_len);
    printf("%s%s\n", padded, deviceProp.computePreemptionSupported ? "Yes" : "No");

    pad_string(padded, "  Device supports Stream Priority:", description_len);
    printf("%s%s\n", padded, deviceProp.streamPrioritiesSupported ? "Yes" : "No");

    pad_string(padded, "  Device supports storing globals in L1 cache:", description_len);
    printf("%s%s\n", padded, deviceProp.globalL1CacheSupported ? "Yes" : "No");

    pad_string(padded, "  Device supports storing locals in L1 cache:", description_len);
    printf("%s%s\n", padded, deviceProp.localL1CacheSupported ? "Yes" : "No");

    pad_string(padded, "  Device shares a unified memory address space with host:", description_len);
    printf("%s%s\n", padded, deviceProp.unifiedAddressing ? "Yes" : "No");

    pad_string(padded, "  Device supports mapping host memory into CUDA address space:", description_len);
    printf("%s%s\n", padded, deviceProp.canMapHostMemory ? "Yes" : "No");

    pad_string(padded, "  Device supports managed memory alllocation:", description_len);
    printf("%s%s\n", padded, deviceProp.managedMemory ? "Yes" : "No");

    pad_string(padded, "  Device allows host to access managed mameory directly:", description_len);
    printf("%s%s\n", padded, deviceProp.directManagedMemAccessFromHost ? "Yes" : "No");

    pad_string(padded, "  Device support concurrent memory copy and kernel execution: ", description_len);
    if (deviceProp.asyncEngineCount == 0) {
        printf("%sNo\n", padded);
    } else if (deviceProp.asyncEngineCount == 1) {
        printf("%sHalf duplex\n", padded);
    } else if (deviceProp.asyncEngineCount == 2) {
        printf("%sFull duplex\n", padded);
    }

    free(padded);
    return;
}

void print_programming_features(hipDeviceProp_t deviceProp) {
    // warpSize,
    // regsPerBlock,
    // maxThreadsPerBlock,
    // maxThreadsDim,
    // maxGridSize,

    // regsPerMultiprocessor,
    // maxThreadsPerMultiProcessor,
    // sharedMemPerMultiprocessor,
    // maxBlocksPerMultiProcessor,

    const int description_len = 68;
    char *padded = (char *)malloc(sizeof(char) * (description_len + 1));

    pad_string(padded, "  Warp size:", description_len);
    printf("%s%d\n", padded, deviceProp.warpSize);

    pad_string(padded, "  Total number of registers available per block:", description_len);
    printf("%s%d\n", padded, deviceProp.regsPerBlock);

    pad_string(padded, "  Maximum number of threads per block:", description_len);
    printf("%s%d\n", padded, deviceProp.maxThreadsPerBlock);

    pad_string(padded, "  Max dimension size of a thread block (x,y,z):", description_len);
    printf("%s(%d, %d, %d)\n", padded, deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);

    pad_string(padded, "  Max dimension size of a grid size    (x,y,z):", description_len);
    printf("%s(%d, %d, %d)\n", padded, deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);

    pad_string(padded, "  Registers per multiprocessor:", description_len);
    printf("%s%d\n", padded, deviceProp.regsPerMultiprocessor);

    pad_string(padded, "  Maximum number of threads per multiprocessor:", description_len);
    printf("%s%d\n", padded, deviceProp.maxThreadsPerMultiProcessor);

    pad_string(padded, "  Total shared memory per multiprocessor:", description_len);
    printf("%s%ld bytes\n", padded, deviceProp.sharedMemPerMultiprocessor);

    pad_string(padded, "  Max blocks per multiprocessor:", description_len);
    printf("%s%d\n", padded, deviceProp.maxBlocksPerMultiProcessor);

    free(padded);
    return;
}

void pad_string(char *des, char *src, size_t target_length) {
    size_t string_length = strlen(src);

    if (string_length > target_length) {
        memcpy(des, src, target_length+1);
        return;
    }

    memset(des, 32, (target_length + 1));
    memcpy(des, src, string_length);
    des[target_length + 1] = '\0';
    return;
}
